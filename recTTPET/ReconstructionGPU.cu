
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>

__global__
void sinogram(int *n, double **x, double **y, double **phi, double **sign,double **rsino,int j)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  //  int j = blockIdx.y*blockDim.y + threadIdx.y;

  //  if(j<m && i<n[j]){
  if(i<n[j]){
 rsino[j][i]=-sqrt(x[j][i]*x[j][i]+y[j][i]*y[j][i])*sign[j][i];

  }

//blockIdx variabile di tipo dim3, contiene l'indice del blocco
//threadIdx variabile di tipo dim3, contiene l'indice del thread
}

int main(void ) {


   const int nslice=1;
   double xmin=-20;
   double xmax=20;
   double zmin=-25;
   double zmax=25;
   double ymin=-20;
   double ymax=20;
   double voxelsize=0.125;
   
   int Nvoxelz=nslice;
   double voxelsizez=(zmax-zmin)/Nvoxelz;
   int Nvoxel=(xmax-xmin)/voxelsize;//512;
   //double **hx=(double**)malloc(nslice*sizeof(double));

  std::vector<double> hx[nslice];
  std::vector<double> hy[nslice];
  std::vector<double> hz[nslice];
  std::vector<double> hphi[nslice];
  std::vector<double> hsign[nslice];


   int idxslice[nslice];

  for(int j=0;j<nslice;j++){
  idxslice[j]=0;
  
  hx[j].reserve(5);
  hy[j].reserve(5);
  hz[j].reserve(5);
  hphi[j].reserve(5);	
  hsign[j].reserve(5);

 
}
const int Nmaxcoinc=10;

  double ene1[Nmaxcoinc],ene2[Nmaxcoinc],deltat[Nmaxcoinc],x[Nmaxcoinc],y[Nmaxcoinc],z[Nmaxcoinc],sign[Nmaxcoinc],xhit1[Nmaxcoinc],yhit1[Nmaxcoinc],zhit1[Nmaxcoinc],xhit2[Nmaxcoinc],yhit2[Nmaxcoinc],\
zhit2[Nmaxcoinc],phi[Nmaxcoinc],theta[Nmaxcoinc];

   int pad1[Nmaxcoinc],pad2[Nmaxcoinc],cell1[Nmaxcoinc],cell2[Nmaxcoinc],chip1[Nmaxcoinc],chip2[Nmaxcoinc],pix1[Nmaxcoinc],pix2[Nmaxcoinc],samecell1[Nmaxcoinc],sharedcharge1[Nmaxcoinc],samecell2[Nmax\
coinc],sharedcharge2[Nmaxcoinc],gammaID1[Nmaxcoinc],triggered,NEntries;


FILE *filein=std::fopen("./binary_10000_12000_0.bin","r");

std::fread(&NEntries,4,1,filein);

for(int i=0;i<10;i++){
 //std::cout<<NEntries<<std::endl;
  if(i%1000==0)std::cout<<i<<std::endl;
  
  std::fread(&(int&)triggered,4,1,filein);
  
  //std::cout<<triggered<<std::endl;
  
  for(int k=0;k<triggered;k++){
    
    
    std::fread(&(double&)ene1[k],8,1,filein);
    std::fread(&(double&)ene2[k],8,1,filein);
    std::fread(&(int&)cell1[k],4,1,filein);
    std::fread(&(int&)cell2[k],4,1,filein);
    std::fread(&(int&)pad1[k],4,1,filein);
    std::fread(&(int&)pad2[k],4,1,filein);
    std::fread(&(int&)chip1[k],4,1,filein);
    std::fread(&(int&)chip2[k],4,1,filein);
    std::fread(&(int&)pix1[k],4,1,filein);
    std::fread(&(int&)pix2[k],4,1,filein);
    std::fread(&(double&)deltat[k],8,1,filein);
    std::fread(&(double&)sign[k],8,1,filein); //for sinogram
    std::fread(&(double&)x[k],8,1,filein);
    std::fread(&(double&)y[k],8,1,filein);
    std::fread(&(double&)z[k],8,1,filein);
    std::fread(&(double&)phi[k],8,1,filein);
    std::fread(&(double&)theta[k],8,1,filein);
    
    
    for(int j=0;j<nslice;j++){
      
      if(z[k]>zmin+j*voxelsizez && z[k]<zmin+(j+1)*voxelsizez){
	
	hz[j].push_back(z[k]);
	hy[j].push_back(y[k]);
	hx[j].push_back(x[k]);
	hphi[j].push_back(phi[k]);	
	hsign[j].push_back(sign[k]);
	idxslice[j]++;
	
      }
      
    }
  }
 }
 
//end of data sorting




 //copy to device	
 
 double *dx[nslice],*dy[nslice],*dz[nslice],*dphi[nslice],*dsign[nslice],*drsino[nslice];//device arrays
 double *hdx[nslice],*hdy[nslice],*hdz[nslice],*hdphi[nslice],*hdsign[nslice],*hdrsino[nslice],*hdx1[nslice];//host arrays for vector dynamic allocation
 
 for(int j=0;j<nslice;j++){


   hdx[j]=(double*)malloc(idxslice[j]*sizeof(double));
   hdx1[j]=(double*)malloc(idxslice[j]*sizeof(double));
   hdy[j]=(double*)malloc(idxslice[j]*sizeof(double));
   hdz[j]=(double*)malloc(idxslice[j]*sizeof(double));
   hdsign[j]=(double*)malloc(idxslice[j]*sizeof(double));
   hdphi[j]=(double*)malloc(idxslice[j]*sizeof(double));
   hdrsino[j]=(double*)malloc(idxslice[j]*sizeof(double));
   
   for(int k=0;k<idxslice[j];k++){
     
     hdx[j][k]=j+k;//hx[j][k];
     hdy[j][k]=hy[j][k];
     hdz[j][k]=hz[j][k];
     hdsign[j][k]=hx[j][k];
     hdphi[j][k]=hx[j][k];

   }
   
   //allocating memory on device
   hipMalloc(&dx[j],idxslice[j]*sizeof(double));
   hipMalloc(&dy[j],idxslice[j]*sizeof(double));
   hipMalloc(&dz[j],idxslice[j]*sizeof(double));
   hipMalloc(&dphi[j],idxslice[j]*sizeof(double));
   hipMalloc(&dsign[j],idxslice[j]*sizeof(double));
   hipMalloc(&drsino[j],idxslice[j]*sizeof(double));  


   //copying to device
   hipMemcpy(dx[j],hdx[j], idxslice[j]*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dy[j],hdy[j], idxslice[j]*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dz[j], hdz[j], idxslice[j]*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dphi[j],hdphi[j], idxslice[j]*sizeof(double), hipMemcpyHostToDevice);
   hipMemcpy(dsign[j],hdsign[j], idxslice[j]*sizeof(double), hipMemcpyHostToDevice);
  
 }

 // for(int j=0;j<nslice;j++) sinogram<<<1,idxslice[j]>>>(idxslice,dx,dy,dphi,dsign,drsino,j);

  for(int j=0;j<nslice;j++){
    hipMemcpy(hdx1[j],dx[j], idxslice[j]*sizeof(double), hipMemcpyDeviceToHost);
    //  cudaMemcpy(hdrsino[j],drsino[j], idxslice[j]*sizeof(double), cudaMemcpyDeviceToHost);
  }
  for(int j=0;j<nslice;j++){
    for(int k=0;k<idxslice[j];k++){
      // std::cout<<"mortaccitua "<<hdrsino[j][k] <<" "<<hdx1[j][k]<<" "<<hy[j][k]<<std::endl;
       std::cout<<"mortaccitua "<<hdx1[j][k]<<" "<<hdx[j][k]<<std::endl;
    }
  }
 
 for(int j=0;j<nslice;j++){
   hipFree(dx[j]);
   hipFree(dy[j]);
   hipFree(dz[j]);
   hipFree(dsign[j]);
   hipFree(dphi[j]);
   hipFree(drsino[j]);
   free(hdrsino[j]);
   free(hdx[j]);
   free(hdx1[j]);
   free(hdy[j]);
   free(hdz[j]);
   free(hdphi[j]);
   free(hdsign[j]);
   hx[j].clear();
   hy[j].clear();
   hz[j].clear();
   hphi[j].clear();
   hsign[j].clear();
   
 }
 
}		
