#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>


__global__
void reco3d(int n, double *r, double *s,double *phi,double *z)
{
  
  // printf("check %d\n",n);
  
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if(i<n){
    
    //  printf("%f %f %f %f\n",r[i],s[i],phi[i],z[i]);

    double a=r[i]*cos(phi[i]);
   
  }
}

__global__
void copy(int *n,int *dummy,int nslice)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  
  if(i<nslice){
    
    dummy[i]=n[i];
    printf("check %d %d\n",i,n[i]);
  }
  
  //blockIdx variabile di tipo dim3, contiene l'indice del blocco
  //threadIdx variabile di tipo dim3, contiene l'indice del thread
}

int main(void ) {
    
  time_t Start, Stop;
  
  const int nslice=100;
  int Nevents=1000000;
  double xmin=-20;
  double xmax=20;
  double zmin=-25;
  double zmax=25;
  double ymin=-20;
  double ymax=20;
  double voxelsize=0.125;
  
  int Nvoxelz=nslice;
  double voxelsizez=(zmax-zmin)/Nvoxelz;
  int Nvoxel=(xmax-xmin)/voxelsize;//512;
  //double **hx=(double**)malloc(nslice*sizeof(double));
  
  
  thrust::host_vector<double> hs[nslice];
  thrust::host_vector<double> hr[nslice];
  thrust::host_vector<double> hz[nslice];
  thrust::host_vector<double> hphi[nslice];
  
  
  int idxslice[nslice];
  
  for(int j=0;j<nslice;j++){
    idxslice[j]=0;
    
    hr[j].reserve(5);
    hs[j].reserve(5);
    hz[j].reserve(5);
    hphi[j].reserve(5);	
 
    
  }
  const int Nmaxcoinc=10;
  
  double ene1[Nmaxcoinc],ene2[Nmaxcoinc],deltat[Nmaxcoinc],x[Nmaxcoinc],y[Nmaxcoinc],z[Nmaxcoinc],sign[Nmaxcoinc],xhit1[Nmaxcoinc],yhit1[Nmaxcoinc],zhit1[Nmaxcoinc],xhit2[Nmaxcoinc],yhit2[Nmaxcoinc],zhit2[Nmaxcoinc],phi[Nmaxcoinc],theta[Nmaxcoinc];
  
  int pad1[Nmaxcoinc],pad2[Nmaxcoinc],cell1[Nmaxcoinc],cell2[Nmaxcoinc],chip1[Nmaxcoinc],chip2[Nmaxcoinc],pix1[Nmaxcoinc],pix2[Nmaxcoinc],samecell1[Nmaxcoinc],sharedcharge1[Nmaxcoinc],samecell2[Nmaxcoinc],sharedcharge2[Nmaxcoinc],gammaID1[Nmaxcoinc],triggered,NEntries;
  
  
  FILE *filein=std::fopen("./binary_10000_11000_0.bin","r");
  
  std::fread(&NEntries,4,1,filein);
  
  for(int i=0;i<Nevents;i++){
    //std::cout<<NEntries<<std::endl;
    if(i%1000==0)std::cout<<i<<std::endl;
    
    std::fread(&(int&)triggered,4,1,filein);
    
    //std::cout<<triggered<<std::endl;
  
    for(int k=0;k<triggered;k++){
      
      std::fread(&(double&)xhit1[k],8,1,filein);
      std::fread(&(double&)xhit2[k],8,1,filein);
      std::fread(&(double&)yhit1[k],8,1,filein);
      std::fread(&(double&)yhit2[k],8,1,filein);
      std::fread(&(double&)ene1[k],8,1,filein);
      std::fread(&(double&)ene2[k],8,1,filein);
      std::fread(&(int&)cell1[k],4,1,filein);
      std::fread(&(int&)cell2[k],4,1,filein);
      std::fread(&(int&)pad1[k],4,1,filein);
      std::fread(&(int&)pad2[k],4,1,filein);
      std::fread(&(int&)chip1[k],4,1,filein);
      std::fread(&(int&)chip2[k],4,1,filein);
      std::fread(&(int&)pix1[k],4,1,filein);
      std::fread(&(int&)pix2[k],4,1,filein);
      std::fread(&(double&)deltat[k],8,1,filein);
      std::fread(&(double&)sign[k],8,1,filein); //for sinogram
      std::fread(&(double&)x[k],8,1,filein);
      std::fread(&(double&)y[k],8,1,filein);
      std::fread(&(double&)z[k],8,1,filein);
      std::fread(&(double&)phi[k],8,1,filein);
      std::fread(&(double&)theta[k],8,1,filein);
      double c=299.792458;//mm/ns
      
      for(int j=0;j<nslice;j++){
      
	if(z[k]>zmin+j*voxelsizez && z[k]<zmin+(j+1)*voxelsizez){

	  double s1=xhit1[k]*cos(phi[k])+yhit1[k]*sin(phi[k]);
	  double s2=xhit2[k]*cos(phi[k])+yhit2[k]*sin(phi[k]);
	  double sm=(s1+s2)/2.;
	  
	  double ssino=sm-(c*deltat[k]*0.5)*sin(theta[k]);
	  hz[j].push_back(z[k]);
	  hs[j].push_back(ssino);
	  hr[j].push_back(-sqrt(x[k]*x[k]+y[k]*y[k])*sign[k]);
	  hphi[j].push_back(phi[k]); 
	  idxslice[j]++;
	}
      }
    }
  }
  
  //end of data sorting
  
  
  
  //copy to device	
  
  
 thrust::device_vector<double> ds[nslice];
 thrust::device_vector<double> dr[nslice];
 thrust::device_vector<double> dz[nslice];
 thrust::device_vector<double> dphi[nslice];
 
 hipEvent_t start;
 hipEvent_t stop;
 checkCudaErrors(hipEventCreate(&start));
 checkCudaErrors(hipEventCreate(&stop));
 
 
 
 for(int j=0;j<nslice;j++){
   
   
   ds[j].reserve(idxslice[j]);
   dr[j].reserve(idxslice[j]);
   dz[j].reserve(idxslice[j]);
   dphi[j].reserve(idxslice[j]);	
   
   ds[j]=hs[j];
   dr[j]=hr[j];
   dz[j]=hz[j];
   dphi[j]=hphi[j];

   
 }
 
 checkCudaErrors(hipEventRecord(start, NULL)); //start
 
 int Nthreads=1000;
 
 double *ps[nslice],*pr[nslice],*pz[nslice],*pphi[nslice];
 
 for(int j=0;j<nslice;j++){
   
   ps[j]=thrust::raw_pointer_cast(ds[j].data());
   pr[j]=thrust::raw_pointer_cast(dr[j].data());
   pz[j]=thrust::raw_pointer_cast(dz[j].data());
   pphi[j]=thrust::raw_pointer_cast(dphi[j].data());

   reco3d<<<(idxslice[j]+(Nthreads-1))/Nthreads,Nthreads>>>(idxslice[j],ps[j],pr[j],pphi[j],pz[j]);
   
   
 }

 
 
 checkCudaErrors(hipEventRecord(stop, NULL));//stop
 checkCudaErrors(hipEventSynchronize(stop));
 float msecTotal = 0.0f;
 checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
 
 
 time(&Start);
 checkCudaErrors(hipEventRecord(start, NULL)); //start

 for(int j=0;j<nslice;j++){

   for(int k=0;k<idxslice[j];k++)double a=hr[j][k]*cos(hphi[j][k]);

 }
   
   checkCudaErrors(hipEventRecord(stop, NULL));//stop
   checkCudaErrors(hipEventSynchronize(stop));
   float msecTotal1 = 0.0f;
 checkCudaErrors(hipEventElapsedTime(&msecTotal1, start, stop));
 time(&Stop);
 printf("Processing time cpu: %d (sec)\n", Stop - Start);
 printf("Processing time cpu: %f (msec)\n", msecTotal1);
 printf("Processing time gpu: %f (msec)\n", msecTotal);

 for(int j=0;j<nslice;j++){
   hr[j].clear();
   hs[j].clear();
   hz[j].clear();
   hphi[j].clear();
   dr[j].clear();
   ds[j].clear();
   dz[j].clear();
   dphi[j].clear();
   }


}	       

