
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>

__global__
void sum(int n, double *x, double *y, double *z)
{
  
 
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  if(i<n){
    z[i]=x[i]+y[i];
    printf("check from function i= %d %f + %f = %f\n",i,x[i],y[i],z[i]);
  }
  //blockIdx variabile di tipo dim3, contiene l'indice del blocco
  //threadIdx variabile di tipo dim3, contiene l'indice del thread
}




int main(void ) {

  const int Nfix=10;//fixed dimension
  double *a[Nfix],*b[Nfix],*c[Nfix]; //for the host
  double *da[Nfix],*db[Nfix],*dc[Nfix]; //for the device

    
  int Nvar[Nfix]; //dynamic dimension
  
  //then I allocate the memory
  
  for(int i=0;i<Nfix;i++){

    Nvar[i]=3*(i+1);
    
    a[i]=(double*)malloc(Nvar[i]*sizeof(double));
    b[i]=(double*)malloc(Nvar[i]*sizeof(double));
    c[i]=(double*)malloc(Nvar[i]*sizeof(double));
    
  

    for(int k=0;k<Nvar[i];k++){
      a[i][k]=k+(i+1);
      b[i][k]=k+2*(i+1);
    } 
    
    //and for the device
    hipMalloc(&da[i],Nvar[i]*sizeof(double));
    hipMalloc(&db[i],Nvar[i]*sizeof(double));
    hipMalloc(&dc[i],Nvar[i]*sizeof(double));
    
    //the I copy the memory 
    
    hipMemcpy(da[i],a[i],Nvar[i]*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(db[i],b[i],Nvar[i]*sizeof(double),hipMemcpyHostToDevice);

    sum<<<1,Nvar[i]>>>(Nvar[i],da[i],db[i],dc[i]);
    hipMemcpy(c[i],dc[i],Nvar[i]*sizeof(double),hipMemcpyDeviceToHost);

    for(int k=0;k<Nvar[i];k++)std::cout<<"i= "<<k<<" "<<a[i][k]<<" + "<<b[i][k]<<" = "<<c[i][k]<<std::endl;

    hipFree(da[i]);
    hipFree(db[i]);
    hipFree(dc[i]);
    free(a[i]);
    free(b[i]);
    free(c[i]);

    
  }
  
}


