
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__
void saxpy(int n, float a, float *x, float *y,float *sum)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) sum[i] = a*x[i] + y[i];

//blockIdx variabile di tipo dim3, contiene l'indice del blocco
//threadIdx variabile di tipo dim3, contiene l'indice del thread
}



int main(void){

  float *hy,*hx,*dx,*dy,*hdsum,*sum;
  int N=100;
  hx=(float*)malloc(N*sizeof(float));
  hy=(float*)malloc(N*sizeof(float));
  hdsum=(float*)malloc(N*sizeof(float));
  hipMalloc(&dx,N*sizeof(float));
  hipMalloc(&dy,N*sizeof(float));
  hipMalloc(&sum,N*sizeof(float));
  
  for(int k=0;k<N;k++){
    hx[k]=1.0f;
    hy[k]=2.0f;
  }
  hipMemcpy(dx,hx, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dy,hy, N*sizeof(float), hipMemcpyHostToDevice);
  
 
  
  saxpy<<<1, N>>>(N, 1.0f,dx,dy,sum);
  
  hipMemcpy(hdsum,sum, N*sizeof(float), hipMemcpyDeviceToHost);
  
  for(int k=0;k<N;k++)std::cout<<hdsum[k]<<" "<<hy[k]<<std::endl;
}
